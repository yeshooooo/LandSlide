#include "hip/hip_runtime.h"
#include "Header.cuh"
#include <iostream>
#include ""
//#include "hip/device_functions.h"
#include <cmath>
#include "hip/hip_runtime_api.h"


using namespace std;
using namespace cv;

__global__ void mergeKernel(cuda::PtrStepSz<float2> out, cuda::PtrStepSz<float2> mat1, cuda::PtrStepSz<float2> mat2)
{
	const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; //������
	//int j = threadIdx.y + blockIdx.y * blockDim.y; //������

	if (i < mat1.cols) // ����Ҫ����Ƿ�Խ��
	{
		out(0, i) = mat1(0, i);
	}
	else
	{
		out(0, i) = mat2(0, i - mat1.cols);
	}
	//__syncthreads();
}

__global__ void siftKernel(cuda::PtrStepSz<float2> out, cuda::PtrStepSz<float2> mat1, cuda::PtrStepSz<float2> mat2, cuda::PtrStepSz<uchar> status, int* index)
{
	const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (1 != status(0, id))
	{
		return;
	}
	if (mat1(0, id).y > mat2(0, id).y)
		return;
	float dx = (mat1(0, id).x - mat2(0, id).x);
	float dy = (mat1(0, id).y - mat2(0, id).y);
	if (3 < dx + dy)
	{
		return;
	}
	out(0, *index) = out(0, id);
	atomicAdd(index, 1);
}

hipError_t callback(cv::cuda::GpuMat &d_better, cv::cuda::GpuMat &d_nuwP) {
	//int *dev_a = nullptr;
	int col = d_nuwP.size().width;
	if (1 != d_nuwP.rows)
	{
		cerr << "sth wrong\n";
		exit(-1);

	}
	//size_t gridSize = 2;
	dim3 blockSize(1, col / 2);
	//mat.operator cuda::PtrStepSz<uchar1>;
	//hipStream_t stream 

	//CE(hipSetDevice(0));

	//CE(hipMalloc((void**)&dev_a, size * sizeof(int)));

	//CE(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

	//size_t blockSize = size / 2;  // block�Ĵ�С��size/2
	//kernel << <gridSize, blockSize >> > (mat);

	//CE(hipGetLastError());
	//CE(hipDeviceSynchronize());

	// ���Դ��а����ݸ��ƻ��ڴ�
	//CE(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

	//CE(hipFree(dev_b));
	//CE(hipDeviceReset());
	return hipError_t();
}

cv::cuda::GpuMat merge(cv::cuda::GpuMat &d_better, cv::cuda::GpuMat &d_nuwP)
{
	int col1 = d_better.cols;
	int col2 = d_nuwP.cols;
	int col = (col1 + col2);
	cuda::GpuMat out(1, col, d_better.type());
	mergeKernel << <1, col>> > (out, d_better, d_nuwP);
	CE(hipGetLastError());
	CE(hipDeviceSynchronize());
	return out;
}

void sift(cv::cuda::GpuMat &corners, cv::cuda::GpuMat &backcross, cv::cuda::GpuMat &nextPtr, cv::cuda::GpuMat &status)
{
	int col = nextPtr.cols;
	//int *index = new int[nextPtr.cols];
	int * index;
	int * tmp = new int;
	CE(hipMalloc((void**)&index, sizeof(int)));
	hipMemset(index, 0, sizeof(int));
	//hipMallocManaged(&index, 0);
	//CE(hipMemset((void**)&index, sizeof(int)));
	siftKernel << <1, col >> > (nextPtr, corners, backcross, status, index);
	CE(hipMemcpy(tmp, index, sizeof(int), hipMemcpyDeviceToHost));
	nextPtr = nextPtr.colRange(0, *tmp);
	CE(hipGetLastError());
	CE(hipDeviceSynchronize());
	CE(hipFree(index));
	delete tmp;
}

